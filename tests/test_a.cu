
#include <stdio.h>
#include "hip/hip_runtime.h"

__device__ void cuda_device_func(int* a) {
  a[0] = 1;
}

__global__ void cuda_entry_point(int* a) {
  cuda_device_func(a);
}

int main() {
  int* a{nullptr};
  hipMalloc(&a, sizeof(int));
  cuda_entry_point<<<1,1,1>>>(a);
  int b{};
  hipMemcpy(&b, &a, sizeof(int), hipMemcpyDefault);
  printf("%s says hello world, result=%d\n", __FILE__, b);
}
