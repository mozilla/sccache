#include <stdio.h>
#include "hip/hip_runtime.h"

__device__ void cuda_device_func(int* a) {
  a[0] = 1;
  a[1] = 2;
}

__global__ void cuda_entry_point(int* a) {
  cuda_device_func(a);
}

int main() {
  int* a;
  hipMalloc(&a, sizeof(int) * 2);
  cuda_entry_point<<<1,1>>>(a);
  int b;
  hipMemcpy(&b, a, sizeof(int), hipMemcpyDeviceToHost);
  printf("%s says hello world, result=%d\n", __FILE__, b);
}
