#include <stdio.h>
#include "hip/hip_runtime.h"

__device__ void cuda_device_func(int* a) {
  #  if __CUDA_ARCH__ < 860
  a[0] = 3;
  #  else
  a[0] = 2;
  #  endif
}

__global__ void cuda_entry_point(int* a) {
  cuda_device_func(a);
}

int main() {
  int* a;
  hipMalloc(&a, sizeof(int));
  cuda_entry_point<<<1,1>>>(a);
  int b;
  hipMemcpy(&b, a, sizeof(int), hipMemcpyDeviceToHost);
  printf("%s says hello world, result=%d\n", __FILE__, b);
}
